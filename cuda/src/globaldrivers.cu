// Convenience routines moved from gptl.cpp to here so that gptl.cpp can be understood
// by nvcc as host-only code.

#include "config.h" /* Must be first include. */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../include/private.h"
#include "../../include/devicehost.h"
#include "../include/gptl_cuda.h"

extern "C" {
  
void GPTLfinalize_gpu_host ()
{
  GPTLfinalize_gpu<<<1,1>>>();
}

void GPTLreset_gpu_host ()
{
  GPTLreset_gpu<<<1,1>>>();
}

// Return useful GPU properties. Use arg list for SMcount, cores_per_sm, and cores_per_gpu even 
// though they're globals, because this is a user-callable routine
int GPTLget_gpu_props (int *khz, int *warpsize, int *devnum, int *SMcount,
		       int *cores_per_sm, int *cores_per_gpu)
{
  hipDeviceProp_t prop;
  size_t size;
  hipError_t err;
  static const size_t onemb = 1024 * 1024;
  static const char *thisfunc = "GPTLget_gpu_props";

  if ((err = hipGetDeviceProperties (&prop, 0)) != hipSuccess) {
    printf ("%s: error:%s", thisfunc, hipGetErrorString (err));
    return -1;
  }

  *khz           = prop.clockRate;
  *warpsize      = prop.warpSize;
  *SMcount       = prop.multiProcessorCount;
#ifdef HAVE_HELPER_CUDA_H
  *cores_per_sm  = _ConvertSMVer2Cores (prop.major, prop.minor);
  *cores_per_gpu = *cores_per_sm * (*SMcount);
#else
  *cores_per_sm  = -1;
  *cores_per_gpu = -1;
#endif  
  printf ("%s: major.minor=%d.%d\n", thisfunc, prop.major, prop.minor);
  printf ("%s: SM count=%d\n",      thisfunc, *SMcount);
  printf ("%s: cores per sm=%d\n",  thisfunc, *cores_per_sm);
  printf ("%s: cores per GPU=%d\n", thisfunc, *cores_per_gpu);

  err = hipGetDevice (devnum);  // device number
  err = hipDeviceGetLimit (&size, hipLimitMallocHeapSize);
  printf ("%s: default hipLimitMallocHeapSize=%d MB\n", thisfunc, (int) (size / onemb));
  return 0;
}

int GPTLcudadevsync (void)
{
  hipDeviceSynchronize ();
  return 0;
}
}
